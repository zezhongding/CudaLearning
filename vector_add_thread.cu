
#include <hip/hip_runtime.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = 0;
    int stride = 1;
    for(int i = index; i < n; i+=stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    hipMalloc((void**)&d_a,sizeof(float)*N);
    
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,256>>>(out, a, b, N);
    hipFree(d_a);
    free(a);
}
